#include "hip/hip_runtime.h"
!pip install git+https://github.com/andreinechaev/nvcc4jupyter.git
%load_ext nvcc_plugin
%%cu
// Exo1: addition de deux vecteurs en mobilisant plusieurs threads
#include <stdio.h>
#include <stdlib.h>
#define N 1000
#define THREAD_PER_BLOCK 512

__global__ void add (int *a , int *b , int *c) 
{  
   int indice = threadIdx.x + blockIdx.x * blockDim.x;
   if (indice < N)
      c[indice] = a[indice] + b[indice]; 
} 

int main (){
   int *a, * b, *c;
   int *gpu_a, *gpu_b, *gpu_c;
   int size = N * sizeof (int); 
      
   hipMalloc ((void **) &gpu_a, size); 
   hipMalloc ((void **) &gpu_b, size);
   hipMalloc ((void **) &gpu_c, size); 
   
   a = (int *) malloc (size); 
   b = (int *) malloc (size); 
   c = (int *) malloc (size); 

   for (int i = 0; i < N; i++)
      {
       a[i] = i;
      }

   for (int i = 0; i < N; i++)
      {
       b[i] = 2*i;
      } 
   
   hipMemcpy (gpu_a, a, size, hipMemcpyHostToDevice); 
   hipMemcpy (gpu_b, b, size, hipMemcpyHostToDevice);
 
   add <<< (N + THREAD_PER_BLOCK) / THREAD_PER_BLOCK, THREAD_PER_BLOCK >>> (gpu_a, gpu_b, gpu_c);

   hipMemcpy (c, gpu_c, size, hipMemcpyDeviceToHost);

   hipFree(gpu_a); hipFree(gpu_b); hipFree(gpu_c);

   for (int i = 0; i < N; i++)
      {
       printf("%d\n", c[i]);
      }

   free(a); free(b); free(c);

   return 0;
}
//------------------------------------------------------------------
%%cu
// Exo2: partie 1, threads appartenant au meme bloc, donc 
// M=N=nbr_threadsTotal=capacitéVecteur et un seul bloc = 1
#include <stdio.h>
#include <stdlib.h>
#define N 512

__global__ void dot (int *a , int *b , int *c) 
{
   __shared__ int temp[N]; 	                                                             
   temp[threadIdx.x] = a[threadIdx.x] * b[threadIdx.x]; 
   
   __syncthreads ();

   // Le thread 0 effectue la somme 
   if (threadIdx.x == 0) {
      int sum = 0;
      for (int i = 0; i < N; i++)
              sum += temp[i];
              *c = sum;        
      }
    } 

int main (){
   int *a, * b, *c;
   int *gpu_a, *gpu_b, *gpu_c;
   int size = N * sizeof (int); 
      
   hipMalloc ((void **) &gpu_a, size); 
   hipMalloc ((void **) &gpu_b, size);
   hipMalloc ((void **) &gpu_c, sizeof (int)); 
   
   a = (int *) malloc (size);
   b = (int *) malloc (size);
   c = (int *) malloc (sizeof (int)); 

   for (int i = 0; i < N; i++)
      {
       a[i] = i;
      }


   for (int i = 0; i < N; i++)
      {
       b[i] = 2*i;
      }

   hipMemcpy (gpu_a, a, size, hipMemcpyHostToDevice); 
   hipMemcpy (gpu_b, b, size, hipMemcpyHostToDevice);
 
   dot <<<1, N>>> (gpu_a, gpu_b, gpu_c);

   hipMemcpy (c, gpu_c, sizeof(int), hipMemcpyDeviceToHost);

   hipFree(gpu_a); hipFree(gpu_b); hipFree(gpu_c);

   printf("%d\n", *c);

   free(a); free(b); free(c);

   return 0;}

//------------------------------------------------------------------
%%cu
// Exo2: partie 2, threads appartenant a déffirents blocs
#include <stdio.h>
#include <stdlib.h>
#define N 2048
#define THREAD_PER_BLOCK 512

__global__ void dot (int *a , int *b , int *c) 
{
   //chaque bloc possède son vecteur temp partager entre les threads du meme bloc
   __shared__ int temp[THREAD_PER_BLOCK];
   int indice = threadIdx.x + blockIdx.x * blockDim.x;
   temp[threadIdx.x] = a[indice] * b[indice];
   __syncthreads ();

   // Le thread 0 de chaque bloc effectue une somme locale qu'il ajoute à la somme globale "atomiquement" 
   if (threadIdx.x == 0) {
      int sum = 0;
	    for (int i = 0; i < THREAD_PER_BLOCK; i++)
           sum += temp[i];
      atomicAdd (c, sum);
   }
} 

int main (){
   int *a, * b, *c;
   int *gpu_a, *gpu_b, *gpu_c;
   int size = N * sizeof (int); 
      
   hipMalloc ((void **) &gpu_a, size); 
   hipMalloc ((void **) &gpu_b, size);
   hipMalloc ((void **) &gpu_c, sizeof (int)); 
   
   a = (int *) malloc (size); 
   b = (int *) malloc (size);
   c = (int *) malloc (sizeof (int)); 

   for (int i = 0; i < N; i++)
      {
       a[i] = i;
      }

   for (int i = 0; i < N; i++)
      {
       b[i] = 2*i;
      }

   hipMemcpy (gpu_a, a, size, hipMemcpyHostToDevice); 
   hipMemcpy (gpu_b, b, size, hipMemcpyHostToDevice);
   
   dot <<<N / THREAD_PER_BLOCK, THREAD_PER_BLOCK >>> (gpu_a, gpu_b, gpu_c);

   hipMemcpy (c, gpu_c, sizeof(int), hipMemcpyDeviceToHost);

   hipFree(gpu_a); hipFree(gpu_b); hipFree(gpu_c);

   printf("%d\n", *c);

   free(a); free(b); free(c);

   return 0;}

//------------------------------------------------------------------
%%cu
// Exo 3 : inverser un vecteur en utilisant differents blocs (matrice logique)
#include <stdio.h>
#include <stdlib.h>
#define N 2048
#define THREAD_PER_BLOCK 512

__global__ void reverseArray (int *d_b , int *d_a)
{
  int old_id = threadIdx.x + blockIdx.x * blockDim.x; 
  int new_id = N - 1 - old_id ; 
  d_b[old_id] = d_a[new_id];
}

int main (){
   int *h_a, *d_a, *d_b;
   int size = N * sizeof (int); 
   h_a = (int *) malloc (size);

   for (int i = 0; i < N; i++)
   {
       h_a[i] = i;
   }

   hipMalloc ((void **) &d_a, size); 
   hipMalloc ((void **) &d_b, size); 
   hipMemcpy (d_a, h_a, size, hipMemcpyHostToDevice);
 
   reverseArray <<< N/THREAD_PER_BLOCK, THREAD_PER_BLOCK >>>(d_b, d_a);

   hipMemcpy (h_a, d_b, size, hipMemcpyDeviceToHost);

   hipFree(d_a); hipFree(d_b); 

   for (int i = 0; i < N; i++)
   {
       printf("%d\n", h_a[i]);
   }


   free(h_a);

   return 0;}

//------------------------------------------------------------------
%%cu
// parallélisation de l'algorithme du calcul nobre pi
#include <stdio.h>
#include <stdlib.h>
#define NUM_BLOCKS 196 
#define NUM_THREADS 512 

__global__ void cal_pi(double *sum, double steps, long nb_steps) 
{
  double x;

  int idx = blockIdx.x * blockDim.x + threadIdx.x; 
 
  if (idx > 0 && idx <= nb_steps)
     x = (idx - 0.5) * steps;
     sum[idx] = 4.0/(1.0 + x*x);
}

int main() 
{

   long nb_steps = 100000;
   double pi = 0.0;
   double steps = 1.0/(double)nb_steps;

   double *sumHost, *sumDev; 
   
   dim3 dimGrid(NUM_BLOCKS,1); 
   dim3 dimBlock(NUM_THREADS,1);

   size_t size = NUM_BLOCKS * NUM_THREADS * sizeof (double); 

   sumHost = (double *) malloc (size);

   hipMalloc ((void **) &sumDev, size); 
   hipMemset (sumDev, 0, size);

   cal_pi <<<dimGrid, dimBlock>>> (sumDev, steps, nb_steps);

   hipMemcpy (sumHost, sumDev, size, hipMemcpyDeviceToHost);

   for(int tid = 1; tid <= nb_steps; tid++)
      pi += sumHost[tid];

   pi *= steps;

   printf("PI=%f\n", pi);

   free (sumHost);

   hipFree (sumDev);

   return 0;}
//------------------------------------------------------------------
%%cu
// Exo 5: multiplication scalaire de deux victeurs
#include <stdio.h>
#include <stdlib.h>
#define BLOCKSIZE 16
#define SIZE 128

__global__ void vectvectshared (int *A, int *B, int *r)
{    
  __shared__ int temp[SIZE];
	
  int i = threadIdx.x; 
  int j = threadIdx.y;

  int ind  = j + (blockDim.x * i);

  if (ind < SIZE)
     temp[ind] = A[ind] * B[ind];
    
   __syncthreads();

   if(ind == 0){
     int sum = 0;
     for(int i = 0; i < SIZE; i++)
        sum += temp[i];
     *r = sum;
   }
}

void fill_dp_vector (int* vec,int size)
{
   int ind;
   for(ind = 0; ind < size; ind++)
        vec[ind] = 3*ind;
}

int main ()
{
   int *hostA, *hostB, *res;
   int *devA, *devB, *devres;

   int vlen;

   vlen=SIZE;
	
   dim3 threadspblock(BLOCKSIZE,BLOCKSIZE);

   hostA = (int *) malloc (vlen * sizeof(int));
   hostB = (int *) malloc (vlen * sizeof(int));
   res = (int *) malloc (sizeof(int));
   
   fill_dp_vector (hostA, vlen);   
   fill_dp_vector (hostB, vlen);

   hipMalloc((void **) &devA, vlen * sizeof(int));
   hipMalloc((void **) &devB, vlen * sizeof(int));
   hipMalloc((void **) &devres, sizeof(int));

   hipMemcpy(devA, hostA, vlen * sizeof(int), hipMemcpyHostToDevice);
   hipMemcpy(devB, hostB, vlen * sizeof(int), hipMemcpyHostToDevice);
		
   vectvectshared<<<1, threadspblock>>>(devA, devB, devres);

   hipMemcpy (res, devres, sizeof(int), hipMemcpyDeviceToHost);
 
   hipFree (devA);
   hipFree (devB);
   hipFree (devres);

   printf("%d\n", *res);

   free (hostA);
   free (hostB);
   free (res);

   return 0;}
